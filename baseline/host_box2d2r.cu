#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <vector>
#include <hip/hip_cooperative_groups.h>
#include "../common/common.hpp"

#define ceil(a, b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

using namespace nvcuda;


// * * * * *
// * * * * *
// * * * * *
// * * * * *
// * * * * *
extern "C" __global__ void kernel_run(half *__restrict__ A, half *__restrict__ C, int N, int tile_size, int *index1, int *index2);


void padding_2D(__half * in, __half * out, int size, int halo_size, __half value) {
    int padding_size = size + halo_size * 2;
    for (int i = 0; i < padding_size; i++) {
        for (int j = 0; j < padding_size; j++) {
            if (halo_size <= i && i < size + halo_size && halo_size <= j && j < size + halo_size) {
                out[i * padding_size + j] = in[(i - halo_size) * size + j - halo_size];
            } else {
                out[i * padding_size + j] = value;
            }
            
        }
    }
}

void brick_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * n + j) * block_size * block_size + x * block_size + y] = in[(i * block_size + x) * size + j * block_size + y];
                }
            }
        }
    }
}

void reverse_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * block_size + x) * size + j * block_size + y] = in[(i * n + j) * block_size * block_size + x * block_size + y];
                }
            }
        }
    }
}

void set_halo_index(int *index1, int *index2, int N) {
    for (int i = 0; i < 4; i++) {
        int y = i % 2;
        int x = i / 2;
        index1[i] = 20 * y + x;
        index2[i] = x * 16 - N * 16 + y - 18;
    }
    for (int i = 4; i < 8; i++) {
        int y = i % 2;
        int x = i / 2;
        index1[i] = 20 * y + x + 16;
        index2[i] = x * 16 + N * 16 + y - 274;
    }
    for (int i = 8; i < 12; i++) {
        int y = i % 2;
        int x = i / 2;
        index1[i] = x + y * 20 + 36;
        index2[i] = x * 16 - N * 16 + y + 416;
    }
    for (int i = 12; i < 16; i++) {
        int y = i % 2;
        int x = i / 2;
        index1[i] = x + y * 20 + 52;
        index2[i] = x * 16 + N * 16 + y + 160;
    }
}

// data size is N * N. Because of halo, we only update (N-4) * (N-4)
extern "C" void host_code(float *h_A, float *h_B, float *h_C, int N, int n) {

    half *A;
    half *in = array_float2half(h_A, N * N);
    half *half_A = new half[(N + 32) * (N + 32)];
    padding_2D(in, half_A, N, 16, __float2half(1.0));
    delete[] in;
    half *half_C = new half[(N + 32) * (N + 32)];

    brick_layout_2D(half_A, half_C, N + 32, 16);

    hipMalloc(&A, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for A");
    hipMemcpy(A, half_C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    half *C;
    hipMalloc(&C, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for C");
    hipMemcpy(C, half_A, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    // halo index set
    int h_index1[16], h_index2[16];
    set_halo_index(h_index1, h_index2, N + 32);
    int *index1, *index2;
    hipMalloc(&index1, sizeof(int) * 16);
    check_error("Failed to allocate device memory for index1");
    hipMalloc(&index2, sizeof(int) * 16);
    check_error("Failed to allocate device memory for index2");
    hipMemcpy(index1, h_index1, sizeof(int) * 16, hipMemcpyHostToDevice);
    hipMemcpy(index2, h_index2, sizeof(int) * 16, hipMemcpyHostToDevice);

    dim3 blockconfig(16, 2);
    int tile_size = TILE_SIZE;
    dim3 gridconfig(ceil(N, 16 * tile_size), ceil(N, 16));

    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int test_t = 0; test_t < RUN_TIMES; test_t++) {
        // hipMemcpy(A, half_A, sizeof(half) * N * N, hipMemcpyHostToDevice);
        half *in = A;
        half *out = C;
        hipDeviceSynchronize();
        hipEventRecord(start, 0);

        kernel_run<<<gridconfig, blockconfig>>>(in, out, N + 32, tile_size, index1, index2);
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        if (test_t >= SKIP_TIMES)
            sum += elapsed;
    }
    printf("[Time] Time used: %f ms\n", sum / (RUN_TIMES - SKIP_TIMES) / STEP_TIMES);
    check_error("finished");

    hipMemcpy(half_A, C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyDeviceToHost);

    reverse_layout_2D(half_A, half_C, N + 32, 16);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_C[i * N + j] = __half2float(half_C[(i + 16) * (N + 32) + j + 16]);
        }
    }

    delete[] half_A;
    delete[] half_C;
    hipFree(A);
    hipFree(C);
}
