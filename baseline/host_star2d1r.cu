#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <vector>
#include <hip/hip_cooperative_groups.h>
#include "../common/common.hpp"

#define ceil(a, b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

using namespace nvcuda;

void padding_2D(__half * in, __half * out, int size, int halo_size, __half value) {
    int padding_size = size + halo_size * 2;
    for (int i = 0; i < padding_size; i++) {
        for (int j = 0; j < padding_size; j++) {
            if (halo_size <= i && i < size + halo_size && halo_size <= j && j < size + halo_size) {
                out[i * padding_size + j] = in[(i - halo_size) * size + j - halo_size];
            } else {
                out[i * padding_size + j] = value;
            }
            
        }
    }
}

void brick_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * n + j) * block_size * block_size + x * block_size + y] = in[(i * block_size + x) * size + j * block_size + y];
                }
            }
        }
    }
}

void reverse_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * block_size + x) * size + j * block_size + y] = in[(i * n + j) * block_size * block_size + x * block_size + y];
                }
            }
        }
    }
}

void set_halo_index(int *index1, int *index2, int N) {
    for (int i = 0; i < 16; i++) {
        int x = i;
        index1[i] = i;
        index2[i] = i - (N << 4) + 240;

        index1[i + 16] = i + 16;
        index2[i + 16] = i + (N << 4);

        index1[i + 32] = i + 32;
        index2[i + 32] = (x << 4) - 241;

        index1[i + 48] = i + 48;
        index2[i + 48] = (x << 4) + 256;
    }
}

//     *
//     *
// * * * * *
//     *
//     *
//处理9点stencil，半径r=2
extern "C" __global__ void kernel_run(half *__restrict__ A, half *__restrict__ C, int N, int tile_size, int *index1, int *index2);

extern "C" void host_code(float *h_A, float *h_B, float *h_C, int N, int n) {
    half *in = array_float2half(h_A, N * N);
    half *half_A = new half[(N + 32) * (N + 32)];
    padding_2D(in, half_A, N, 16, __float2half(1.0));
    delete[] in;
    half *half_C = new half[(N + 32) * (N + 32)];
    
    brick_layout_2D(half_A, half_C, N + 32, 16);

    half *A;
    hipMalloc(&A, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for A");
    hipMemcpy(A, half_C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    half *C;
    hipMalloc(&C, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for C");
    hipMemcpy(C, half_A, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    int h_index1[64], h_index2[64];
    set_halo_index(h_index1, h_index2, N + 32);
    int *index1, *index2;
    hipMalloc(&index1, sizeof(int) * 64);
    check_error("Failed to allocate device memory for index1");
    hipMalloc(&index2, sizeof(int) * 64);
    check_error("Failed to allocate device memory for index2");
    hipMemcpy(index1, h_index1, sizeof(int) * 64, hipMemcpyHostToDevice);
    hipMemcpy(index2, h_index2, sizeof(int) * 64, hipMemcpyHostToDevice);

    int tile_size = TILE_SIZE;
    dim3 gridconfig(ceil(N, 16 * tile_size), ceil(N, 16));
    dim3 blockconfig(16, 2);

    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum = 0.0;


    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int test_t = 0; test_t < RUN_TIMES; test_t++) {
        // hipMemcpy(A, half_A, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);
        half *in = A;
        half *out = C;
        hipDeviceSynchronize();
        hipEventRecord(start, 0);

        kernel_run<<<gridconfig, blockconfig>>>(in, out, N + 32, tile_size, index1, index2);
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        if (test_t >= SKIP_TIMES)
            sum += elapsed;
    }
    printf("[Time] Time used: %f ms\n", sum / (RUN_TIMES - SKIP_TIMES) / STEP_TIMES);
    check_error("finished");

    hipMemcpy(half_A, C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyDeviceToHost);

    reverse_layout_2D(half_A, half_C, N + 32, 16);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_C[i * N + j] = __half2float(half_C[(i + 16) * (N + 32) + j + 16]);
        }
    }

    delete[] half_A;
    delete[] half_C;
    hipFree(A);
    hipFree(C);
    hipFree(index1);
    hipFree(index2);
}
