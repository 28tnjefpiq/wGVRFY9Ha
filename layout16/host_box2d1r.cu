#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <vector>
#include <hip/hip_cooperative_groups.h>
#include "../common/common.hpp"

#define ceil(a, b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

using namespace nvcuda;


// coe
// * - - - - - - - *
// | * * * * * . . |
// | * * * * * . . |
// | * * * * * . . |
// | * * * * * . . |
// | * * * * * . . |
// | . * * * * * . |
// | . * * * * * . |
// | . * * * * * . |
// | . * * * * * . |
// | . * * * * * . |
// | . . . . . . . |
// * - - - - - - - *
void init_coe(half *__restrict__ B, half *__restrict__ coe, int n) {
    for (int i = 0; i < 16 * 16 * 3; i++) {
        coe[i] = __float2half(0);
    }
    for(int z = 0; z < 3; z++){
        int offset = z * 256;
        for(int i = 0; i < 14; i++){
            for(int j = 0; j < 3; j++){
                coe[offset + 17 * i + 16 * j + 1] = B[z * 3 + j];
            }
        }
        // full part coe
        for (int i = 0; i < 2; i++) {
            coe[offset + i * 16] = B[z * 3 + i + 1];
            coe[offset + i * 16 + 239] = B[z * 3 + i];
        }
    }
}

// * * * * *
// * * * * *
// * * * * *
// * * * * *
// * * * * *
extern "C" __global__ void mma_run(half *__restrict__ A, half *__restrict__ coe,
                               half *__restrict__ C, int N, int tile_size, int *index1, int *index2);


void padding_2D(__half * in, __half * out, int size, int halo_size, __half value) {
    int padding_size = size + halo_size * 2;
    for (int i = 0; i < padding_size; i++) {
        for (int j = 0; j < padding_size; j++) {
            if (halo_size <= i && i < size + halo_size && halo_size <= j && j < size + halo_size) {
                out[i * padding_size + j] = in[(i - halo_size) * size + j - halo_size];
            } else {
                out[i * padding_size + j] = value;
            }
            
        }
    }
}

void brick_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * n + j) * block_size * block_size + x * block_size + y] = in[(i * block_size + x) * size + j * block_size + y];
                }
            }
        }
    }
}

void reverse_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * block_size + x) * size + j * block_size + y] = in[(i * n + j) * block_size * block_size + x * block_size + y];
                }
            }
        }
    }
}

void set_halo_index(int *index1, int *index2, int N) {
    index1[0] = 0;
    index1[1] = 17;
    index1[2] = 18;
    index1[3] = 35;

    index2[0] = -N * 16 - 1;
    index2[1] = N * 16 - 256 + 15;
    index2[2] = -N * 16 + 240;
    index2[3] = N * 16 + 256;
}

// data size is N * N. Because of halo, we only update (N-4) * (N-4)
extern "C" void host_code(float *h_A, float *h_B, float *h_C, int N, int n) {
    half h_coe[16 * 16 * 3];
    half *half_B = array_float2half(h_B, 9);
    init_coe(half_B, h_coe, n);

    half *A, *coe;
    half *in = array_float2half(h_A, N * N);
    half *half_A = new half[(N + 32) * (N + 32)];
    padding_2D(in, half_A, N, 16, __float2half(1.0));
    delete[] in;
    half *half_C = new half[(N + 32) * (N + 32)];

    brick_layout_2D(half_A, half_C, N + 32, 16);

    hipMalloc(&A, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for A");
    hipMemcpy(A, half_C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    hipMalloc(&coe, sizeof(half) * 16 * 16 * 3);
    check_error("Failed to allocate device memory for coe");
    hipMemcpy(coe, h_coe, sizeof(half) * 16 * 16 * 3, hipMemcpyHostToDevice);

    half *C;
    hipMalloc(&C, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for C");
    hipMemcpy(C, half_A, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    // halo index set
    int h_index1[4], h_index2[4];
    set_halo_index(h_index1, h_index2, N + 32);
    int *index1, *index2;
    hipMalloc(&index1, sizeof(int) * 4);
    check_error("Failed to allocate device memory for index1");
    hipMalloc(&index2, sizeof(int) * 4);
    check_error("Failed to allocate device memory for index2");
    hipMemcpy(index1, h_index1, sizeof(int) * 4, hipMemcpyHostToDevice);
    hipMemcpy(index2, h_index2, sizeof(int) * 4, hipMemcpyHostToDevice);

    dim3 blockconfig(16, 2);
    int tile_size = TILE_SIZE;
    dim3 gridconfig(ceil(N, 16 * tile_size), ceil(N, 16));

    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int test_t = 0; test_t < RUN_TIMES; test_t++) {
        // hipMemcpy(A, half_A, sizeof(half) * N * N, hipMemcpyHostToDevice);
        half *in = A;
        half *out = C;
        hipDeviceSynchronize();
        hipEventRecord(start, 0);

        mma_run<<<gridconfig, blockconfig>>>(in, coe, out, N + 32, tile_size, index1, index2);
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        if (test_t >= SKIP_TIMES)
            sum += elapsed;
    }
    printf("[Time] Time used: %f ms\n", sum / (RUN_TIMES - SKIP_TIMES) / STEP_TIMES);
    check_error("finished");

    hipMemcpy(half_A, C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyDeviceToHost);

    reverse_layout_2D(half_A, half_C, N + 32, 16);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_C[i * N + j] = __half2float(half_C[(i + 16) * (N + 32) + j + 16]);
        }
    }

    delete[] half_A;
    delete[] half_B;
    delete[] half_C;
    hipFree(A);
    hipFree(coe);
    hipFree(C);
}
