#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <vector>
#include "../common/common.hpp"

#define ceil(a, b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

using namespace nvcuda;

// a
// * - - - - - - - *
// | * * * * * . . |
// | . * * * * * . |
// | . . * * * * * |
// * - - - - - - - *
// b
// * - - - - - - - *
// | * . . . . . . |
// | * * . . . . . |
// | * * * . . . . |
// | * * * * . . . |
// | * * * * * . . |
// | . * * * * * . |
// | . . * * * * * |
// | . . . * * * * |
// | . . . . * * * |
// | . . . . . * * |
// | . . . . . . * |
// * - - - - - - - *
void init_coe(half *__restrict__ B, half *__restrict__ a, half *__restrict__ b,
              int n) {
    for (int i = 0; i < 16; i++) {
        for (int j = 0; j < 16; j++) {
            a[i * 16 + j] = __float2half(0);
            b[i * 16 + j] = __float2half(0);
        }
    }
    half vertical_buf[3], horizon_buf[3];
    for (int i = 0; i < 3; i++) {
        horizon_buf[i] = B[n + i];
        vertical_buf[i] = B[1 + i * n];
    }
    horizon_buf[1] = __float2half(0);

    for (int i = 0; i < 14; i++) {
        for (int j = 0; j < 3; j++) {
            a[i * 16 + i + j + 16] = vertical_buf[j];
            b[i * 16 + i + j * 16 + 1] = horizon_buf[j];
        }
    }
    // full part coe
    for (int i = 0; i < 2; i++) {
        a[i] = vertical_buf[i + 1];
        a[i + 254] = vertical_buf[i];
        b[i * 16] = horizon_buf[i + 1];
        b[i * 16 + 15 + 224] = horizon_buf[i];
    }
}

void padding_2D(__half * in, __half * out, int size, int halo_size, __half value) {
    int padding_size = size + halo_size * 2;
    for (int i = 0; i < padding_size; i++) {
        for (int j = 0; j < padding_size; j++) {
            if (halo_size <= i && i < size + halo_size && halo_size <= j && j < size + halo_size) {
                out[i * padding_size + j] = in[(i - halo_size) * size + j - halo_size];
            } else {
                out[i * padding_size + j] = value;
            }
            
        }
    }
}

void brick_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * n + j) * block_size * block_size + x * block_size + y] = in[(i * block_size + x) * size + j * block_size + y];
                }
            }
        }
    }
}

void reverse_layout_2D(__half * in, __half * out, int size, int block_size) {
    int n = size / block_size;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int x = 0; x < block_size; x++) {
                for (int y = 0; y < block_size; y++) {
                    out[(i * block_size + x) * size + j * block_size + y] = in[(i * n + j) * block_size * block_size + x * block_size + y];
                }
            }
        }
    }
}

void set_halo_index(int *index1, int *index2, int N) {
    for (int i = 0; i < 16; i++) {
        int x = i;
        index1[i] = i;
        index2[i] = i - (N << 4) + 240;

        index1[i + 16] = i + 16;
        index2[i + 16] = i + (N << 4);

        index1[i + 32] = i + 32;
        index2[i + 32] = (x << 4) - 241;

        index1[i + 48] = i + 48;
        index2[i + 48] = (x << 4) + 256;
    }
}

//     *
//     *
// * * * * *
//     *
//     *
extern "C" __global__ void mma_run(half *A, half *__restrict__ coe_a,
                               half *__restrict__ coe_b, half *C,
                               int N, int tile_size, int * index1, int * index2);

extern "C" void host_code(float *h_A, float *h_B, float *h_C, int N, int n) {
    half h_coe_a[16 * 16], h_coe_b[16 * 16];
    half *half_B = array_float2half(h_B, 9);
    init_coe(half_B, h_coe_a, h_coe_b, n);

    int h_index1[64], h_index2[64];
    set_halo_index(h_index1, h_index2, N + 32);

    half *A, *coe_a, *coe_b;
    half *in = array_float2half(h_A, N * N);
    half *half_A = new half[(N + 32) * (N + 32)];
    padding_2D(in, half_A, N, 16, __float2half(1.0));
    delete[] in;
    half *half_C = new half[(N + 32) * (N + 32)];
    
    brick_layout_2D(half_A, half_C, N + 32, 16);

    hipMalloc(&A, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for A");
    hipMemcpy(A, half_C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    hipMalloc(&coe_a, sizeof(half) * 16 * 16);
    check_error("Failed to allocate device memory for coe_a");
    hipMemcpy(coe_a, h_coe_a, sizeof(half) * 16 * 16, hipMemcpyHostToDevice);

    hipMalloc(&coe_b, sizeof(half) * 16 * 16);
    check_error("Failed to allocate device memory for coe_b");
    hipMemcpy(coe_b, h_coe_b, sizeof(half) * 16 * 16, hipMemcpyHostToDevice);

    half *C;
    hipMalloc(&C, sizeof(half) * (N + 32) * (N + 32));
    check_error("Failed to allocate device memory for C");
    hipMemcpy(C, half_A, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);

    int *index1, *index2;
    hipMalloc(&index1, sizeof(int) * 64);
    check_error("Failed to allocate device memory for index1");
    hipMalloc(&index2, sizeof(int) * 64);
    check_error("Failed to allocate device memory for index2");
    hipMemcpy(index1, h_index1, sizeof(int) * 64, hipMemcpyHostToDevice);
    hipMemcpy(index2, h_index2, sizeof(int) * 64, hipMemcpyHostToDevice);
    

    int tile_size = TILE_SIZE;
    dim3 gridconfig(ceil(N, 16 * tile_size), ceil(N, 16));
    dim3 blockconfig(16, 2);

    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum = 0.0;


    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int test_t = 0; test_t < RUN_TIMES; test_t++) {
        // hipMemcpy(A, half_A, sizeof(half) * (N + 32) * (N + 32), hipMemcpyHostToDevice);
        half *in = A;
        half *out = C;
        hipDeviceSynchronize();
        hipEventRecord(start, 0);

        mma_run<<<gridconfig, blockconfig>>>(in, coe_a, coe_b, out, N + 32, tile_size, index1, index2);
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        if (test_t >= SKIP_TIMES)
            sum += elapsed;
    }
    printf("[Time] Time used: %f ms\n", sum / (RUN_TIMES - SKIP_TIMES) / STEP_TIMES);
    check_error("finished");

    hipMemcpy(half_A, C, sizeof(half) * (N + 32) * (N + 32), hipMemcpyDeviceToHost);

    reverse_layout_2D(half_A, half_C, N + 32, 16);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_C[i * N + j] = __half2float(half_C[(i + 16) * (N + 32) + j + 16]);
        }
    }

    delete[] half_A;
    delete[] half_B;
    delete[] half_C;
    hipFree(A);
    hipFree(coe_a);
    hipFree(coe_b);
    hipFree(C);
    hipFree(index1);
    hipFree(index2);
}
