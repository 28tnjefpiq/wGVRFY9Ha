#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <vector>
#include "../common/common.hpp"

#define ceil(a, b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

using namespace nvcuda;

//     *
//     *
// * * * * *
//     *
//     *
extern "C" __global__ void mma_run(half *__restrict__ A, half *__restrict__ coe_a, half *__restrict__ coe_b, half *__restrict__ C, int N, int tile_size, int *index1, int *index2) {
    __shared__ half data[256];
    __shared__ half halo[64];
    const int index = threadIdx.x + (threadIdx.y << 4);
    const int offset_base = (blockIdx.y + 1) * (N << 4) + ((blockIdx.x * tile_size + 1) << 8);
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag[2];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag[2];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> c_frag;
    wmma::load_matrix_sync(a_frag[0], coe_a, 16);
    wmma::load_matrix_sync(b_frag[1], coe_b, 16);

    #pragma unroll
    for(int i = 0; i < tile_size; i++){
        int offset = offset_base + (i << 8);

        halo[index] = A[offset + index2[index]];
        halo[index + 32] = A[offset + index2[index + 32]];

        wmma::load_matrix_sync(b_frag[0], A + offset, 16);
        wmma::load_matrix_sync(a_frag[1], A + offset, 16);
        wmma::fill_fragment(c_frag, 0.0f);
        wmma::mma_sync(c_frag, a_frag[0], b_frag[0], c_frag);
        wmma::mma_sync(c_frag, a_frag[1], b_frag[1], c_frag);
        wmma::store_matrix_sync(data, c_frag, 16, wmma::mem_row_major);
        // do halo compute
        // top and left
        if (threadIdx.y == 0) {
            data[threadIdx.x] += halo[threadIdx.x] * __float2half(1);

            data[index2[index + 32] + 241] += halo[threadIdx.x + 32] * __float2half(3);
        } // bottom and right
        else {
            data[threadIdx.x + 240] += halo[threadIdx.x + 16] * __float2half(7);

            data[index2[index + 16] + 256] += halo[threadIdx.x + 48] * __float2half(5);
        }
        __syncthreads();
        // write
        ((float4 *)(C + offset))[index] = ((float4 *)data)[index];
        // #pragma unroll
        // for (int j = 0; j < 8; j++){
        //     C[index + offset + (j << 5)] = data[index + (j << 5)];
        // }
    }
}
